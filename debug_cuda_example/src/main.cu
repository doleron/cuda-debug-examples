#include <iostream>

void print_CUDA_version()
{
    std::cout
        << "__CUDACC_VER_MAJOR__: " << __CUDACC_VER_MAJOR__ << "\n"
        << "__CUDACC_VER_MINOR__" << __CUDACC_VER_MINOR__ << "\n"
        << "__CUDACC_VER_BUILD__: " << __CUDACC_VER_BUILD__ << "\n\n";

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << "\n";

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << "\n";
}

int main()
{
    std::cout << "Running cudas.\n";

    print_CUDA_version();

    return 0;
}